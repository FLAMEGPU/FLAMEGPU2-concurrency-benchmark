#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <chrono>

#include "flamegpu/flamegpu.h"

/**
 * FLAME GPU 2 concurrency benchmark, using an implementation of the Boids flocking model.
 */

/**
 * Get the length of a vector
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @return the length of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
    return sqrtf(x * x + y * y + z * z);
}

/**
 * Add a scalar to a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param value scalar value to add
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
    x += value;
    y += value;
    z += value;
}

/**
 * Subtract a scalar from a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param value scalar value to subtract
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
    x -= value;
    y -= value;
    z -= value;
}

/**
 * Multiply a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param multiplier scalar value to multiply by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
    x *= multiplier;
    y *= multiplier;
    z *= multiplier;
}

/**
 * Divide a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param divisor scalar value to divide by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
    x /= divisor;
    y /= divisor;
    z /= divisor;
}

/**
 * Normalize a 3 component vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
    // Get the length
    float length = vec3Length(x, y, z);
    vec3Div(x, y, z, length);
}

/**
 * Clamp each component of a 3-part position to lie within a minimum and maximum value.
 * Performs the operation in place
 * Unlike the FLAME GPU 1 example, this is a clamping operation, rather than wrapping.
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param MIN_POSITION the minimum value for each component
 * @param MAX_POSITION the maximum value for each component
 */
FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
    x = (x < MIN_POSITION)? MIN_POSITION: x;
    x = (x > MAX_POSITION)? MAX_POSITION: x;

    y = (y < MIN_POSITION)? MIN_POSITION: y;
    y = (y > MAX_POSITION)? MAX_POSITION: y;

    z = (z < MIN_POSITION)? MIN_POSITION: z;
    z = (z > MAX_POSITION)? MAX_POSITION: z;
}

/**
 * outputdata agent function for Boid agents, which outputs publicly visible properties to a message list
 */
const char* outputdata = R"###(
FLAMEGPU_AGENT_FUNCTION(outputdata, flamegpu::MessageNone, flamegpu::MessageSpatial3D) {
    // Output each agents publicly visible properties.
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    FLAMEGPU->message_out.setVariable<float>("fx", FLAMEGPU->getVariable<float>("fx"));
    FLAMEGPU->message_out.setVariable<float>("fy", FLAMEGPU->getVariable<float>("fy"));
    FLAMEGPU->message_out.setVariable<float>("fz", FLAMEGPU->getVariable<float>("fz"));
    return flamegpu::ALIVE;
}
)###";

const char* outputdataBruteForce = R"###(
    FLAMEGPU_AGENT_FUNCTION(outputdata, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
        // Output each agents publicly visible properties.
        FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
        FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
        FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
        FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
        FLAMEGPU->message_out.setVariable<float>("fx", FLAMEGPU->getVariable<float>("fx"));
        FLAMEGPU->message_out.setVariable<float>("fy", FLAMEGPU->getVariable<float>("fy"));
        FLAMEGPU->message_out.setVariable<float>("fz", FLAMEGPU->getVariable<float>("fz"));
        return flamegpu::ALIVE;
    }
    )###";
/**
 * inputdata agent function for Boid agents, which reads data from neighbouring Boid agents, to perform the boid flocking model.
 */
const char* inputdata = R"###(
// Vector utility functions, see top of file for versions with commentary
FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
    return sqrtf(x * x + y * y + z * z);
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
    x += value;
    y += value;
    z += value;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
    x -= value;
    y -= value;
    z -= value;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
    x *= multiplier;
    y *= multiplier;
    z *= multiplier;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
    x /= divisor;
    y /= divisor;
    z /= divisor;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
    // Get the length
    float length = vec3Length(x, y, z);
    vec3Div(x, y, z, length);
}
FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
    x = (x < MIN_POSITION)? MIN_POSITION: x;
    x = (x > MAX_POSITION)? MAX_POSITION: x;

    y = (y < MIN_POSITION)? MIN_POSITION: y;
    y = (y > MAX_POSITION)? MAX_POSITION: y;

    z = (z < MIN_POSITION)? MIN_POSITION: z;
    z = (z > MAX_POSITION)? MAX_POSITION: z;
}
// Agent function
FLAMEGPU_AGENT_FUNCTION(inputdata, flamegpu::MessageSpatial3D, flamegpu::MessageNone) {
    // Agent properties in local register
    const flamegpu::id_t id = FLAMEGPU->getID();
    // Agent position
    float agent_x = FLAMEGPU->getVariable<float>("x");
    float agent_y = FLAMEGPU->getVariable<float>("y");
    float agent_z = FLAMEGPU->getVariable<float>("z");
    // Agent velocity
    float agent_fx = FLAMEGPU->getVariable<float>("fx");
    float agent_fy = FLAMEGPU->getVariable<float>("fy");
    float agent_fz = FLAMEGPU->getVariable<float>("fz");

    // Boids percieved center
    float perceived_centre_x = 0.0f;
    float perceived_centre_y = 0.0f;
    float perceived_centre_z = 0.0f;
    int perceived_count = 0;

    // Boids global velocity matching
    float global_velocity_x = 0.0f;
    float global_velocity_y = 0.0f;
    float global_velocity_z = 0.0f;

    // Total change in velocity
    float velocity_change_x = 0.f;
    float velocity_change_y = 0.f;
    float velocity_change_z = 0.f;

    const float INTERACTION_RADIUS = FLAMEGPU->environment.getProperty<float>("INTERACTION_RADIUS");
    const float SEPARATION_RADIUS = FLAMEGPU->environment.getProperty<float>("SEPARATION_RADIUS");
    // Iterate location messages, accumulating relevant data and counts.
    for (const auto &message : FLAMEGPU->message_in(agent_x, agent_y, agent_z)) {
        // Ignore self messages.
        if (message.getVariable<flamegpu::id_t>("id") != id) {
            // Get the message location and velocity.
            const float message_x = message.getVariable<float>("x");
            const float message_y = message.getVariable<float>("y");
            const float message_z = message.getVariable<float>("z");

            // Check interaction radius
            float separation = vec3Length(agent_x - message_x, agent_y - message_y, agent_z - message_z);

            if (separation < INTERACTION_RADIUS) {
                // Update the percieved centre
                perceived_centre_x += message_x;
                perceived_centre_y += message_y;
                perceived_centre_z += message_z;
                perceived_count++;

                // Update percieved velocity matching
                const float message_fx = message.getVariable<float>("fx");
                const float message_fy = message.getVariable<float>("fy");
                const float message_fz = message.getVariable<float>("fz");
                global_velocity_x += message_fx;
                global_velocity_y += message_fy;
                global_velocity_z += message_fz;

                // Update collision centre
                if (separation < (SEPARATION_RADIUS)) {  // dependant on model size
                    // Rule 3) Avoid other nearby boids (Separation)
                    float normalizedSeparation = (separation / SEPARATION_RADIUS);
                    float invNormSep = (1.0f - normalizedSeparation);
                    float invSqSep = invNormSep * invNormSep;

                    const float collisionScale = FLAMEGPU->environment.getProperty<float>("COLLISION_SCALE");
                    velocity_change_x += collisionScale * (agent_x - message_x) * invSqSep;
                    velocity_change_y += collisionScale * (agent_y - message_y) * invSqSep;
                    velocity_change_z += collisionScale * (agent_z - message_z) * invSqSep;
                }
            }
        }
    }

    if (perceived_count) {
        // Divide positions/velocities by relevant counts.
        vec3Div(perceived_centre_x, perceived_centre_y, perceived_centre_z, perceived_count);
        vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, perceived_count);

        // Rule 1) Steer towards perceived centre of flock (Cohesion)
        float steer_velocity_x = 0.f;
        float steer_velocity_y = 0.f;
        float steer_velocity_z = 0.f;

        const float STEER_SCALE = FLAMEGPU->environment.getProperty<float>("STEER_SCALE");
        steer_velocity_x = (perceived_centre_x - agent_x) * STEER_SCALE;
        steer_velocity_y = (perceived_centre_y - agent_y) * STEER_SCALE;
        steer_velocity_z = (perceived_centre_z - agent_z) * STEER_SCALE;

        velocity_change_x += steer_velocity_x;
        velocity_change_y += steer_velocity_y;
        velocity_change_z += steer_velocity_z;

        // Rule 2) Match neighbours speeds (Alignment)
        float match_velocity_x = 0.f;
        float match_velocity_y = 0.f;
        float match_velocity_z = 0.f;

        const float MATCH_SCALE = FLAMEGPU->environment.getProperty<float>("MATCH_SCALE");
        match_velocity_x = global_velocity_x;
        match_velocity_y = global_velocity_y;
        match_velocity_z = global_velocity_z;

        velocity_change_x += (match_velocity_x - agent_fx) * MATCH_SCALE;
        velocity_change_y += (match_velocity_y - agent_fy) * MATCH_SCALE;
        velocity_change_z += (match_velocity_z - agent_fz) * MATCH_SCALE;
    }

    // Global scale of velocity change
    vec3Mult(velocity_change_x, velocity_change_y, velocity_change_z, FLAMEGPU->environment.getProperty<float>("GLOBAL_SCALE"));

    // Update agent velocity
    agent_fx += velocity_change_x;
    agent_fy += velocity_change_y;
    agent_fz += velocity_change_z;

    // Bound velocity
    float agent_fscale = vec3Length(agent_fx, agent_fy, agent_fz);
    if (agent_fscale > 1) {
        vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);
    }

    float minSpeed = 0.5f;
    if (agent_fscale < minSpeed) {
        // Normalise
        vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);

        // Scale to min
        vec3Mult(agent_fx, agent_fy, agent_fz, minSpeed);
    }

    // Steer away from walls - Computed post normalization to ensure good avoidance. Prevents constant term getting swamped
    const float wallInteractionDistance = 0.10f;
    const float wallSteerStrength = 0.05f;
    const float minPosition = FLAMEGPU->environment.getProperty<float>("MIN_POSITION");
    const float maxPosition = FLAMEGPU->environment.getProperty<float>("MAX_POSITION");

    if (agent_x - minPosition < wallInteractionDistance) {
        agent_fx += wallSteerStrength;
    }
    if (agent_y - minPosition < wallInteractionDistance) {
        agent_fy += wallSteerStrength;
    }
    if (agent_z - minPosition < wallInteractionDistance) {
        agent_fz += wallSteerStrength;
    }

    if (maxPosition - agent_x < wallInteractionDistance) {
        agent_fx -= wallSteerStrength;
    }
    if (maxPosition - agent_y < wallInteractionDistance) {
        agent_fy -= wallSteerStrength;
    }
    if (maxPosition - agent_z < wallInteractionDistance) {
        agent_fz -= wallSteerStrength;
    }

    // Apply the velocity
    const float TIME_SCALE = FLAMEGPU->environment.getProperty<float>("TIME_SCALE");
    agent_x += agent_fx * TIME_SCALE;
    agent_y += agent_fy * TIME_SCALE;
    agent_z += agent_fz * TIME_SCALE;

    // Bound position
    clampPosition(agent_x, agent_y, agent_z, FLAMEGPU->environment.getProperty<float>("MIN_POSITION"), FLAMEGPU->environment.getProperty<float>("MAX_POSITION"));

    // Update global agent memory.
    FLAMEGPU->setVariable<float>("x", agent_x);
    FLAMEGPU->setVariable<float>("y", agent_y);
    FLAMEGPU->setVariable<float>("z", agent_z);

    FLAMEGPU->setVariable<float>("fx", agent_fx);
    FLAMEGPU->setVariable<float>("fy", agent_fy);
    FLAMEGPU->setVariable<float>("fz", agent_fz);

    return flamegpu::ALIVE;
}
)###";

const char* inputdataBruteForce = R"###(
    // Vector utility functions, see top of file for versions with commentary
    FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
        return sqrtf(x * x + y * y + z * z);
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
        x += value;
        y += value;
        z += value;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
        x -= value;
        y -= value;
        z -= value;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
        x *= multiplier;
        y *= multiplier;
        z *= multiplier;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
        x /= divisor;
        y /= divisor;
        z /= divisor;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
        // Get the length
        float length = vec3Length(x, y, z);
        vec3Div(x, y, z, length);
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
        x = (x < MIN_POSITION)? MIN_POSITION: x;
        x = (x > MAX_POSITION)? MAX_POSITION: x;
    
        y = (y < MIN_POSITION)? MIN_POSITION: y;
        y = (y > MAX_POSITION)? MAX_POSITION: y;
    
        z = (z < MIN_POSITION)? MIN_POSITION: z;
        z = (z > MAX_POSITION)? MAX_POSITION: z;
    }
    // Agent function
    FLAMEGPU_AGENT_FUNCTION(inputdata, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    // Agent properties in local register
    const flamegpu::id_t id = FLAMEGPU->getID();
    // Agent position
    float agent_x = FLAMEGPU->getVariable<float>("x");
    float agent_y = FLAMEGPU->getVariable<float>("y");
    float agent_z = FLAMEGPU->getVariable<float>("z");
    // Agent velocity
    float agent_fx = FLAMEGPU->getVariable<float>("fx");
    float agent_fy = FLAMEGPU->getVariable<float>("fy");
    float agent_fz = FLAMEGPU->getVariable<float>("fz");

    // Boids percieved center
    float perceived_centre_x = 0.0f;
    float perceived_centre_y = 0.0f;
    float perceived_centre_z = 0.0f;
    int perceived_count = 0;

    // Boids global velocity matching
    float global_velocity_x = 0.0f;
    float global_velocity_y = 0.0f;
    float global_velocity_z = 0.0f;

    // Total change in velocity
    float velocity_change_x = 0.f;
    float velocity_change_y = 0.f;
    float velocity_change_z = 0.f;

    const float INTERACTION_RADIUS = FLAMEGPU->environment.getProperty<float>("INTERACTION_RADIUS");
    const float SEPARATION_RADIUS = FLAMEGPU->environment.getProperty<float>("SEPARATION_RADIUS");
    // Iterate location messages, accumulating relevant data and counts.
    for (const auto &message : FLAMEGPU->message_in) {
        // Ignore self messages.
        if (message.getVariable<flamegpu::id_t>("id") != id) {
            // Get the message location and velocity.
            const float message_x = message.getVariable<float>("x");
            const float message_y = message.getVariable<float>("y");
            const float message_z = message.getVariable<float>("z");

            // Check interaction radius
            float separation = vec3Length(agent_x - message_x, agent_y - message_y, agent_z - message_z);

            if (separation < INTERACTION_RADIUS) {
                // Update the percieved centre
                perceived_centre_x += message_x;
                perceived_centre_y += message_y;
                perceived_centre_z += message_z;
                perceived_count++;

                // Update percieved velocity matching
                const float message_fx = message.getVariable<float>("fx");
                const float message_fy = message.getVariable<float>("fy");
                const float message_fz = message.getVariable<float>("fz");
                global_velocity_x += message_fx;
                global_velocity_y += message_fy;
                global_velocity_z += message_fz;

                // Update collision centre
                if (separation < (SEPARATION_RADIUS)) {  // dependant on model size
                    // Rule 3) Avoid other nearby boids (Separation)
                    float normalizedSeparation = (separation / SEPARATION_RADIUS);
                    float invNormSep = (1.0f - normalizedSeparation);
                    float invSqSep = invNormSep * invNormSep;

                    const float collisionScale = FLAMEGPU->environment.getProperty<float>("COLLISION_SCALE");
                    velocity_change_x += collisionScale * (agent_x - message_x) * invSqSep;
                    velocity_change_y += collisionScale * (agent_y - message_y) * invSqSep;
                    velocity_change_z += collisionScale * (agent_z - message_z) * invSqSep;
                }
            }
        }
    }

    if (perceived_count) {
        // Divide positions/velocities by relevant counts.
        vec3Div(perceived_centre_x, perceived_centre_y, perceived_centre_z, perceived_count);
        vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, perceived_count);

        // Rule 1) Steer towards perceived centre of flock (Cohesion)
        float steer_velocity_x = 0.f;
        float steer_velocity_y = 0.f;
        float steer_velocity_z = 0.f;

        const float STEER_SCALE = FLAMEGPU->environment.getProperty<float>("STEER_SCALE");
        steer_velocity_x = (perceived_centre_x - agent_x) * STEER_SCALE;
        steer_velocity_y = (perceived_centre_y - agent_y) * STEER_SCALE;
        steer_velocity_z = (perceived_centre_z - agent_z) * STEER_SCALE;

        velocity_change_x += steer_velocity_x;
        velocity_change_y += steer_velocity_y;
        velocity_change_z += steer_velocity_z;

        // Rule 2) Match neighbours speeds (Alignment)
        float match_velocity_x = 0.f;
        float match_velocity_y = 0.f;
        float match_velocity_z = 0.f;

        const float MATCH_SCALE = FLAMEGPU->environment.getProperty<float>("MATCH_SCALE");
        match_velocity_x = global_velocity_x;
        match_velocity_y = global_velocity_y;
        match_velocity_z = global_velocity_z;

        velocity_change_x += (match_velocity_x - agent_fx) * MATCH_SCALE;
        velocity_change_y += (match_velocity_y - agent_fy) * MATCH_SCALE;
        velocity_change_z += (match_velocity_z - agent_fz) * MATCH_SCALE;
    }

    // Global scale of velocity change
    vec3Mult(velocity_change_x, velocity_change_y, velocity_change_z, FLAMEGPU->environment.getProperty<float>("GLOBAL_SCALE"));

    // Update agent velocity
    agent_fx += velocity_change_x;
    agent_fy += velocity_change_y;
    agent_fz += velocity_change_z;

    // Bound velocity
    float agent_fscale = vec3Length(agent_fx, agent_fy, agent_fz);
    if (agent_fscale > 1) {
        vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);
    }

    float minSpeed = 0.5f;
    if (agent_fscale < minSpeed) {
        // Normalise
        vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);

        // Scale to min
        vec3Mult(agent_fx, agent_fy, agent_fz, minSpeed);
    }

    // Steer away from walls - Computed post normalization to ensure good avoidance. Prevents constant term getting swamped
    const float wallInteractionDistance = 0.10f;
    const float wallSteerStrength = 0.05f;
    const float minPosition = FLAMEGPU->environment.getProperty<float>("MIN_POSITION");
    const float maxPosition = FLAMEGPU->environment.getProperty<float>("MAX_POSITION");

    if (agent_x - minPosition < wallInteractionDistance) {
        agent_fx += wallSteerStrength;
    }
    if (agent_y - minPosition < wallInteractionDistance) {
        agent_fy += wallSteerStrength;
    }
    if (agent_z - minPosition < wallInteractionDistance) {
        agent_fz += wallSteerStrength;
    }

    if (maxPosition - agent_x < wallInteractionDistance) {
        agent_fx -= wallSteerStrength;
    }
    if (maxPosition - agent_y < wallInteractionDistance) {
        agent_fy -= wallSteerStrength;
    }
    if (maxPosition - agent_z < wallInteractionDistance) {
        agent_fz -= wallSteerStrength;
    }

    // Apply the velocity
    const float TIME_SCALE = FLAMEGPU->environment.getProperty<float>("TIME_SCALE");
    agent_x += agent_fx * TIME_SCALE;
    agent_y += agent_fy * TIME_SCALE;
    agent_z += agent_fz * TIME_SCALE;

    // Bound position
    clampPosition(agent_x, agent_y, agent_z, FLAMEGPU->environment.getProperty<float>("MIN_POSITION"), FLAMEGPU->environment.getProperty<float>("MAX_POSITION"));

    // Update global agent memory.
    FLAMEGPU->setVariable<float>("x", agent_x);
    FLAMEGPU->setVariable<float>("y", agent_y);
    FLAMEGPU->setVariable<float>("z", agent_z);

    FLAMEGPU->setVariable<float>("fx", agent_fx);
    FLAMEGPU->setVariable<float>("fy", agent_fy);
    FLAMEGPU->setVariable<float>("fz", agent_fz);

    return flamegpu::ALIVE;
    }
    )###";

typedef struct Experiment { 

    Experiment(std::string title, 
        unsigned int initialPopSize, unsigned int finalPopSize, unsigned int popSizeIncrement, 
        unsigned int initialNumSpecies, unsigned int finalNumSpecies, unsigned int numSpeciesIncrement,
        unsigned int repetitions,
        double targetPopDensity,
        bool spatial,
        unsigned int steps) {
            this->title = title;
            this->initialPopSize = initialPopSize;
            this->finalPopSize = finalPopSize;
            this->popSizeIncrement = popSizeIncrement;
            this->initialNumSpecies = initialNumSpecies;
            this->finalNumSpecies = finalNumSpecies;
            this->numSpeciesIncrement = numSpeciesIncrement;
            this->repetitions = repetitions;
            this->targetPopDensity = targetPopDensity;
            this->spatial = spatial;
            this->steps = steps;
        }
    std::string title;
    unsigned int initialPopSize = 100000;
    unsigned int finalPopSize = 100000;
    unsigned int popSizeIncrement = 100000;
    
    unsigned int initialNumSpecies = 1;
    unsigned int finalNumSpecies = 8;
    unsigned int numSpeciesIncrement = 1;

    unsigned int repetitions = 1;
    double targetPopDensity = 1024;
    bool spatial = true;
    unsigned int steps = 1000;
} Experiment;

int main(int argc, const char ** argv) {

    constexpr unsigned int repetitions = 3;
    Experiment smallFixedPop("Small_Fixed_Pop", 512, 512, 512, 1, 32, 1, repetitions, 1024, true, 1000);
    Experiment smallPops("Small_Pops", 128, 1024, 128, 1, 32, 1, repetitions, 1024, true, 1000);
    Experiment largePops("Large_Pops", 1024, 8192, 1024, 1, 32, 1, repetitions, 1024, true, 1000);
    Experiment deviceMaxed("Device_Maxed", 100000, 100000, 100000, 1, 8, 1, repetitions, 4096, true, 1000);

    Experiment smallFixedPopBruteForce("Small_Fixed_Pop_Brute_Force", 512, 512, 512, 1, 32, 1, repetitions, 1024, false, 1000);
    Experiment smallPopsBruteForce("Small_Pops_Brute_Force", 128, 1024, 128, 1, 32, 1, repetitions, 1024, false, 1000);
    Experiment largePopsBruteForce("Large_Pops_Brute_Force", 1024, 8192, 1024, 1, 32, 1, repetitions, 1024, false, 1000);

    Experiment largePopsFalloff("Large_Pops_Falloff", 2048, 8192, 2048, 1, 127, 8, repetitions, 1024, true, 100);
    Experiment largePopsFalloffBruteForce("Large_Pops_Falloff_Brute_Force", 2048, 8192, 2048, 1, 127, 8, repetitions, 1024, false, 100);

    // std::vector<Experiment> experiments = {largePopsFalloffBruteForce};
    std::vector<Experiment> experiments = {smallFixedPop, smallPops, largePops, deviceMaxed, smallFixedPopBruteForce, smallPopsBruteForce, largePopsBruteForce, largePopsFalloff, largePopsFalloffBruteForce};
    
    for (Experiment experiment : experiments) {
        
        std::cout << std::endl << "Starting experiment: " << experiment.title << std::endl;

        unsigned int initialPopSize = experiment.initialPopSize;
        unsigned int finalPopSize = experiment.finalPopSize;
        unsigned int popSizeIncrement = experiment.popSizeIncrement;
        
        unsigned int initialNumSpecies = experiment.initialNumSpecies;
        unsigned int finalNumSpecies = experiment.finalNumSpecies;
        unsigned int numSpeciesIncrement = experiment.numSpeciesIncrement;

        // Target population density per cubic unit
        double targetPopDensity = experiment.targetPopDensity;

        std::vector<double> serialResults;
        std::vector<double> concurrentResults;
        for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
            for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                serialResults.push_back(0.0);
                concurrentResults.push_back(0.0);
            }
        }

        // Pandas
        std::string csvFileName = experiment.title + ".csv";
        std::ofstream csv(csvFileName, std::ios::trunc);
        csv << "is_concurrent,repetition,pop_size,num_species,s_step_mean" << std::endl;
        
        for (unsigned int isConcurrent = 0; isConcurrent <= 1; isConcurrent++) {
            for (unsigned int repetition = 0; repetition < experiment.repetitions; repetition++) {
                unsigned int resultsIndex = 0;
                for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
                    for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                        std::cout << "Staring run with popSize: " << popSize << ", species: " << numSpecies << std::endl;
                        flamegpu::ModelDescription model("Boids_Concurrency");

                        /**
                        * GLOBALS
                        */
                        flamegpu::EnvironmentDescription env = model.Environment();
                        std::vector<unsigned int> populationSizes;
                        for (unsigned int pops = 0; pops < numSpecies; pops++) {
                            populationSizes.push_back(popSize);
                        }
                        {                          
                            // Target pop density
                            double targetVolume = (double)popSize / targetPopDensity;
                            double sideLength = std::cbrt(targetVolume);
                            double halfSideLength = sideLength / 2.0;
                            //std::cout << "Using side length: " << sideLength << std::endl;


                            // Environment Bounds
                            env.newProperty("MIN_POSITION", static_cast<float>(-halfSideLength));
                            env.newProperty("MAX_POSITION", static_cast<float>(halfSideLength));

                            // Initialisation parameter(s)
                            env.newProperty("MAX_INITIAL_SPEED", 1.0f);
                            env.newProperty("MIN_INITIAL_SPEED", 0.01f);

                            // Interaction radius
                            env.newProperty("INTERACTION_RADIUS", 0.1f);
                            env.newProperty("SEPARATION_RADIUS", 0.005f);

                            // Global Scalers
                            env.newProperty("TIME_SCALE", 0.0005f);
                            env.newProperty("GLOBAL_SCALE", 0.15f);

                            // Rule scalers
                            env.newProperty("STEER_SCALE", 0.65f);
                            env.newProperty("COLLISION_SCALE", 0.75f);
                            env.newProperty("MATCH_SCALE", 1.25f);
                        }

                        {   // Location message
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string messageName = "location";
                                messageName += std::to_string(i);
                                if (experiment.spatial) {
                                    flamegpu::MessageSpatial3D::Description message = model.newMessage<flamegpu::MessageSpatial3D>(messageName);
                                    // Set the range and bounds.
                                    message.setRadius(env.getProperty<float>("INTERACTION_RADIUS"));
                                    message.setMin(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MIN_POSITION"));
                                    message.setMax(env.getProperty<float>("MAX_POSITION"), env.getProperty<float>("MAX_POSITION"), env.getProperty<float>("MAX_POSITION"));
                                    // A message to hold the location of an agent.
                                    message.newVariable<int>("id");
                                    // X Y Z are implicit.
                                    // message.newVariable<float>("x");
                                    // message.newVariable<float>("y");
                                    // message.newVariable<float>("z");
                                    message.newVariable<float>("fx");
                                    message.newVariable<float>("fy");
                                    message.newVariable<float>("fz");
                                } else {
                                    flamegpu::MessageBruteForce::Description message = model.newMessage<flamegpu::MessageBruteForce>(messageName);
                                    // A message to hold the location of an agent.
                                    message.newVariable<int>("id");
                                    message.newVariable<float>("x");
                                    message.newVariable<float>("y");
                                    message.newVariable<float>("z");
                                    message.newVariable<float>("fx");
                                    message.newVariable<float>("fy");
                                    message.newVariable<float>("fz");
                                }
                                
                            }
                        }
                        for (unsigned int i = 0; i < populationSizes.size(); i++) {
                            {   // Boid agent
                                std::string agentName("Boid" + std::to_string(i));
                                flamegpu::AgentDescription agent = model.newAgent(agentName);
                                agent.newVariable<int>("id");
                                agent.newVariable<float>("x");
                                agent.newVariable<float>("y");
                                agent.newVariable<float>("z");
                                agent.newVariable<float>("fx");
                                agent.newVariable<float>("fy");
                                agent.newVariable<float>("fz");
                                std::string messageName = "location";
                                messageName += std::to_string(i);
                                std::string outputFuncName = "outputdata";
                                std::string inputFuncName = "inputdata";
                                
                                // Use brute force or spatial?
                                const char* outputFuncSource;
                                const char* inputFuncSource;
                                if (experiment.spatial) {
                                    outputFuncSource = outputdata;
                                    inputFuncSource = inputdata;
                                } else {
                                    outputFuncSource = outputdataBruteForce;
                                    inputFuncSource = inputdataBruteForce;
                                }
                                agent.newRTCFunction(outputFuncName, outputFuncSource).setMessageOutput(messageName);
                                agent.newRTCFunction(inputFuncName, inputFuncSource).setMessageInput(messageName);
                            }
                        }

                        /**
                        * Control flow
                        */     
                        {   // Layer #1
                            flamegpu::LayerDescription layer = model.newLayer();
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                std::string outputFuncName = "outputdata";
                                layer.addAgentFunction(agentName, outputFuncName);
                            }
                        }
                        {   // Layer #2
                            flamegpu::LayerDescription layer = model.newLayer();
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                std::string inputFuncName = "inputdata";
                                layer.addAgentFunction(agentName, inputFuncName);
                            }
                        }


                        /**
                        * Create Model Runner
                        */
                        flamegpu::CUDASimulation simulation(model);

                        /**
                        * Create visualisation
                        */

                        // Initialisation
                        simulation.initialise(argc, argv);
                        // Set the rng seed to be the current repetition. Cannot currently support overriding via cli due to initailise() implementation in alpha.2
                        simulation.SimulationConfig().random_seed = repetition;
                        // Disable telemetry
                        simulation.SimulationConfig().telemetry = false;

                        // If no xml model file was is provided, generate a population.
                        if (simulation.getSimulationConfig().input_file.empty()) {
                            // Set number of steps
                            simulation.SimulationConfig().steps = experiment.steps;

                            // Uniformly distribute agents within space, with uniformly distributed initial velocity.
                            std::mt19937_64 rngEngine(simulation.getSimulationConfig().random_seed);
                            std::uniform_real_distribution<float> position_distribution(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MAX_POSITION"));
                            std::uniform_real_distribution<float> velocity_distribution(-1, 1);
                            std::uniform_real_distribution<float> velocity_magnitude_distribution(env.getProperty<float>("MIN_INITIAL_SPEED"), env.getProperty<float>("MAX_INITIAL_SPEED"));
                            
                            unsigned int agentCounter = 0;
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                flamegpu::AgentVector population(model.Agent(agentName), populationSizes[i]);
                                for (unsigned int j = 0; j < populationSizes[i]; j++) {
                                    flamegpu::AgentVector::Agent instance = population[i];
                                    instance.setVariable<int>("id", agentCounter);
                                    agentCounter++;

                                    // Agent position in space
                                    instance.setVariable<float>("x", position_distribution(rngEngine));
                                    instance.setVariable<float>("y", position_distribution(rngEngine));
                                    instance.setVariable<float>("z", position_distribution(rngEngine));

                                    // Generate a random velocity direction
                                    float fx = velocity_distribution(rngEngine);
                                    float fy = velocity_distribution(rngEngine);
                                    float fz = velocity_distribution(rngEngine);
                                    // Generate a random speed between 0 and the maximum initial speed
                                    float fmagnitude = velocity_magnitude_distribution(rngEngine);
                                    // Use the random speed for the velocity.
                                    vec3Normalize(fx, fy, fz);
                                    vec3Mult(fx, fy, fz, fmagnitude);

                                    // Set these for the agent.
                                    instance.setVariable<float>("fx", fx);
                                    instance.setVariable<float>("fy", fy);
                                    instance.setVariable<float>("fz", fz);
                                }
                                simulation.setPopulationData(population);
                            }
                        }

                        /**
                        * Execution
                        */
                        simulation.CUDAConfig().inLayerConcurrency = isConcurrent;
                        //std::cout << "In layer concurrency set to: " << simulation.CUDAConfig().inLayerConcurrency << std::endl;

                        simulation.simulate();
                        const auto runTime = simulation.getElapsedTimeSimulation();
                        const double averageStepTime = runTime / static_cast<double>(experiment.steps);
                        
                        //std::cout << "Run complete. Average step time: " << averageStepTime << "S" << std::endl;
                        if (isConcurrent) {
                            concurrentResults[resultsIndex] += averageStepTime;
                        } else {
                            serialResults[resultsIndex] += averageStepTime;
                        }
                        resultsIndex++;

                        //csv << "is_concurrent,repetition,pop_size,num_species,s_step_mean" << std::endl;
                        csv << isConcurrent << "," << repetition << "," << popSize << "," << numSpecies << "," << averageStepTime << std::endl;
                    }
                }
            }
        }
    }
    return 0;
}


