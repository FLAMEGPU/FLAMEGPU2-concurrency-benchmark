#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <chrono>

#include "flamegpu/flame_api.h"

/**
 * FLAME GPU 2 implementation of the Boids model, using spatial3D messaging.
 * This is based on the FLAME GPU 1 implementation, but with dynamic generation of agents. 
 * Agents are also clamped to be within the environment bounds, rather than wrapped as in FLAME GPU 1.
 * 
 * @todo - Should the agent's velocity change when it is clamped to the environment?
 */


/**
 * Get the length of a vector
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @return the length of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
    return sqrtf(x * x + y * y + z * z);
}

/**
 * Add a scalar to a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param value scalar value to add
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
    x += value;
    y += value;
    z += value;
}

/**
 * Subtract a scalar from a vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param value scalar value to subtract
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
    x -= value;
    y -= value;
    z -= value;
}

/**
 * Multiply a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param multiplier scalar value to multiply by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
    x *= multiplier;
    y *= multiplier;
    z *= multiplier;
}

/**
 * Divide a vector by a scalar value in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param divisor scalar value to divide by
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
    x /= divisor;
    y /= divisor;
    z /= divisor;
}

/**
 * Normalize a 3 component vector in-place
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 */ 
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
    // Get the length
    float length = vec3Length(x, y, z);
    vec3Div(x, y, z, length);
}

/**
 * Clamp each component of a 3-part position to lie within a minimum and maximum value.
 * Performs the operation in place
 * Unlike the FLAME GPU 1 example, this is a clamping operation, rather than wrapping.
 * @param x x component of the vector
 * @param y y component of the vector
 * @param z z component of the vector
 * @param MIN_POSITION the minimum value for each component
 * @param MAX_POSITION the maximum value for each component
 */
FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
    x = (x < MIN_POSITION)? MIN_POSITION: x;
    x = (x > MAX_POSITION)? MAX_POSITION: x;

    y = (y < MIN_POSITION)? MIN_POSITION: y;
    y = (y > MAX_POSITION)? MAX_POSITION: y;

    z = (z < MIN_POSITION)? MIN_POSITION: z;
    z = (z > MAX_POSITION)? MAX_POSITION: z;
}



/**
 * outputdata agent function for Boid agents, which outputs publicly visible properties to a message list
 */
const char* outputdata = R"###(
FLAMEGPU_AGENT_FUNCTION(outputdata, MsgNone, MsgSpatial3D) {
    // Output each agents publicly visible properties.
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    FLAMEGPU->message_out.setVariable<float>("fx", FLAMEGPU->getVariable<float>("fx"));
    FLAMEGPU->message_out.setVariable<float>("fy", FLAMEGPU->getVariable<float>("fy"));
    FLAMEGPU->message_out.setVariable<float>("fz", FLAMEGPU->getVariable<float>("fz"));
    return ALIVE;
}
)###";

const char* outputdataBruteForce = R"###(
    FLAMEGPU_AGENT_FUNCTION(outputdata, MsgNone, MsgBruteForce) {
        // Output each agents publicly visible properties.
        FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
        FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
        FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
        FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
        FLAMEGPU->message_out.setVariable<float>("fx", FLAMEGPU->getVariable<float>("fx"));
        FLAMEGPU->message_out.setVariable<float>("fy", FLAMEGPU->getVariable<float>("fy"));
        FLAMEGPU->message_out.setVariable<float>("fz", FLAMEGPU->getVariable<float>("fz"));
        return ALIVE;
    }
    )###";
/**
 * inputdata agent function for Boid agents, which reads data from neighbouring Boid agents, to perform the boid flocking model.
 */
const char* inputdata = R"###(
// Vector utility functions, see top of file for versions with commentary
FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
    return sqrtf(x * x + y * y + z * z);
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
    x += value;
    y += value;
    z += value;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
    x -= value;
    y -= value;
    z -= value;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
    x *= multiplier;
    y *= multiplier;
    z *= multiplier;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
    x /= divisor;
    y /= divisor;
    z /= divisor;
}
FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
    // Get the length
    float length = vec3Length(x, y, z);
    vec3Div(x, y, z, length);
}
FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
    x = (x < MIN_POSITION)? MIN_POSITION: x;
    x = (x > MAX_POSITION)? MAX_POSITION: x;

    y = (y < MIN_POSITION)? MIN_POSITION: y;
    y = (y > MAX_POSITION)? MAX_POSITION: y;

    z = (z < MIN_POSITION)? MIN_POSITION: z;
    z = (z > MAX_POSITION)? MAX_POSITION: z;
}
// Agent function
FLAMEGPU_AGENT_FUNCTION(inputdata, MsgSpatial3D, MsgNone) {

    // Agent properties in local register
    int id = FLAMEGPU->getVariable<int>("id");
    // Agent position
    float agent_x = FLAMEGPU->getVariable<float>("x");
    float agent_y = FLAMEGPU->getVariable<float>("y");
    float agent_z = FLAMEGPU->getVariable<float>("z");
    // Agent velocity
    float agent_fx = FLAMEGPU->getVariable<float>("fx");
    float agent_fy = FLAMEGPU->getVariable<float>("fy");
    float agent_fz = FLAMEGPU->getVariable<float>("fz");

    // Boids percieved center
    float perceived_centre_x = 0.0f;
    float perceived_centre_y = 0.0f;
    float perceived_centre_z = 0.0f;
    int perceived_count = 0;

    // Boids global velocity matching
    float global_velocity_x = 0.0f;
    float global_velocity_y = 0.0f;
    float global_velocity_z = 0.0f;

    // Boids short range avoidance centre
    float collision_centre_x = 0.0f;
    float collision_centre_y = 0.0f;
    float collision_centre_z = 0.0f;
    int collision_count = 0;

    const float INTERACTION_RADIUS = FLAMEGPU->environment.getProperty<float>("INTERACTION_RADIUS");
    const float SEPARATION_RADIUS = FLAMEGPU->environment.getProperty<float>("SEPARATION_RADIUS");
    // Iterate location messages, accumulating relevant data and counts.
    for (const auto &message : FLAMEGPU->message_in(agent_x, agent_y, agent_z)) {
        // Ignore self messages.
        if (message.getVariable<int>("id") != id) {
            // Get the message location and velocity.
            const float message_x = message.getVariable<float>("x");
            const float message_y = message.getVariable<float>("y");
            const float message_z = message.getVariable<float>("z");
            const float message_fx = message.getVariable<float>("fx");
            const float message_fy = message.getVariable<float>("fy");
            const float message_fz = message.getVariable<float>("fz");

            // Check interaction radius
            float separation = vec3Length(agent_x - message_x, agent_y - message_y, agent_z - message_z);

            if (separation < (INTERACTION_RADIUS)) {
                // Update the percieved centre
                perceived_centre_x += message_x;
                perceived_centre_y += message_y;
                perceived_centre_z += message_z;
                perceived_count++;

                // Update percieved velocity matching
                global_velocity_x += message_fx;
                global_velocity_y += message_fy;
                global_velocity_z += message_fz;

                // Update collision centre
                if (separation < (SEPARATION_RADIUS)) {  // dependant on model size
                    collision_centre_x += message_x;
                    collision_centre_y += message_y;
                    collision_centre_z += message_z;
                    collision_count += 1;
                }
            }
        }
    }

    // Divide positions/velocities by relevant counts.
    vec3Div(perceived_centre_x, perceived_centre_y, perceived_centre_z, perceived_count);
    vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, perceived_count);
    vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, collision_count);

    // Total change in velocity
    float velocity_change_x = 0.f;
    float velocity_change_y = 0.f;
    float velocity_change_z = 0.f;

    // Rule 1) Steer towards perceived centre of flock (Cohesion)
    float steer_velocity_x = 0.f;
    float steer_velocity_y = 0.f;
    float steer_velocity_z = 0.f;
    if (perceived_count > 0) {
        const float STEER_SCALE = FLAMEGPU->environment.getProperty<float>("STEER_SCALE");
        steer_velocity_x = (perceived_centre_x - agent_x) * STEER_SCALE;
        steer_velocity_y = (perceived_centre_y - agent_y) * STEER_SCALE;
        steer_velocity_z = (perceived_centre_z - agent_z) * STEER_SCALE;
    }
    velocity_change_x += steer_velocity_x;
    velocity_change_y += steer_velocity_y;
    velocity_change_z += steer_velocity_z;

    // Rule 2) Match neighbours speeds (Alignment)
    float match_velocity_x = 0.f;
    float match_velocity_y = 0.f;
    float match_velocity_z = 0.f;
    if (collision_count > 0) {
        const float MATCH_SCALE = FLAMEGPU->environment.getProperty<float>("MATCH_SCALE");
        match_velocity_x = global_velocity_x * MATCH_SCALE;
        match_velocity_y = global_velocity_y * MATCH_SCALE;
        match_velocity_z = global_velocity_z * MATCH_SCALE;
    }
    velocity_change_x += match_velocity_x;
    velocity_change_y += match_velocity_y;
    velocity_change_z += match_velocity_z;

    // Rule 3) Avoid close range neighbours (Separation)
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;
    float avoid_velocity_z = 0.0f;
    if (collision_count > 0) {
        const float COLLISION_SCALE = FLAMEGPU->environment.getProperty<float>("COLLISION_SCALE");
        avoid_velocity_x = (agent_x - collision_centre_x) * COLLISION_SCALE;
        avoid_velocity_y = (agent_y - collision_centre_y) * COLLISION_SCALE;
        avoid_velocity_z = (agent_z - collision_centre_z) * COLLISION_SCALE;
    }
    velocity_change_x += avoid_velocity_x;
    velocity_change_y += avoid_velocity_y;
    velocity_change_z += avoid_velocity_z;

    // Global scale of velocity change
    vec3Mult(velocity_change_x, velocity_change_y, velocity_change_z, FLAMEGPU->environment.getProperty<float>("GLOBAL_SCALE"));

    // Update agent velocity
    agent_fx += velocity_change_x;
    agent_fy += velocity_change_y;
    agent_fz += velocity_change_z;

    // Bound velocity
    float agent_fscale = vec3Length(agent_fx, agent_fy, agent_fz);
    if (agent_fscale > 1) {
        vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);
    }

    // Apply the velocity
    const float TIME_SCALE = FLAMEGPU->environment.getProperty<float>("TIME_SCALE");
    agent_x += agent_fx * TIME_SCALE;
    agent_y += agent_fy * TIME_SCALE;
    agent_z += agent_fz * TIME_SCALE;

    // Bound position
    clampPosition(agent_x, agent_y, agent_z, FLAMEGPU->environment.getProperty<float>("MIN_POSITION"), FLAMEGPU->environment.getProperty<float>("MAX_POSITION"));

    // Update global agent memory.
    FLAMEGPU->setVariable<float>("x", agent_x);
    FLAMEGPU->setVariable<float>("y", agent_y);
    FLAMEGPU->setVariable<float>("z", agent_z);

    FLAMEGPU->setVariable<float>("fx", agent_fx);
    FLAMEGPU->setVariable<float>("fy", agent_fy);
    FLAMEGPU->setVariable<float>("fz", agent_fz);

    return ALIVE;
}
)###";

const char* inputdataBruteForce = R"###(
    // Vector utility functions, see top of file for versions with commentary
    FLAMEGPU_HOST_DEVICE_FUNCTION float vec3Length(const float x, const float y, const float z) {
        return sqrtf(x * x + y * y + z * z);
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Add(float &x, float &y, float &z, const float value) {
        x += value;
        y += value;
        z += value;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Sub(float &x, float &y, float &z, const float value) {
        x -= value;
        y -= value;
        z -= value;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Mult(float &x, float &y, float &z, const float multiplier) {
        x *= multiplier;
        y *= multiplier;
        z *= multiplier;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Div(float &x, float &y, float &z, const float divisor) {
        x /= divisor;
        y /= divisor;
        z /= divisor;
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void vec3Normalize(float &x, float &y, float &z) {
        // Get the length
        float length = vec3Length(x, y, z);
        vec3Div(x, y, z, length);
    }
    FLAMEGPU_HOST_DEVICE_FUNCTION void clampPosition(float &x, float &y, float &z, const float MIN_POSITION, const float MAX_POSITION) {
        x = (x < MIN_POSITION)? MIN_POSITION: x;
        x = (x > MAX_POSITION)? MAX_POSITION: x;
    
        y = (y < MIN_POSITION)? MIN_POSITION: y;
        y = (y > MAX_POSITION)? MAX_POSITION: y;
    
        z = (z < MIN_POSITION)? MIN_POSITION: z;
        z = (z > MAX_POSITION)? MAX_POSITION: z;
    }
    // Agent function
    FLAMEGPU_AGENT_FUNCTION(inputdata, MsgBruteForce, MsgNone) {
    
        // Agent properties in local register
        int id = FLAMEGPU->getVariable<int>("id");
        // Agent position
        float agent_x = FLAMEGPU->getVariable<float>("x");
        float agent_y = FLAMEGPU->getVariable<float>("y");
        float agent_z = FLAMEGPU->getVariable<float>("z");
        // Agent velocity
        float agent_fx = FLAMEGPU->getVariable<float>("fx");
        float agent_fy = FLAMEGPU->getVariable<float>("fy");
        float agent_fz = FLAMEGPU->getVariable<float>("fz");
    
        // Boids percieved center
        float perceived_centre_x = 0.0f;
        float perceived_centre_y = 0.0f;
        float perceived_centre_z = 0.0f;
        int perceived_count = 0;
    
        // Boids global velocity matching
        float global_velocity_x = 0.0f;
        float global_velocity_y = 0.0f;
        float global_velocity_z = 0.0f;
    
        // Boids short range avoidance centre
        float collision_centre_x = 0.0f;
        float collision_centre_y = 0.0f;
        float collision_centre_z = 0.0f;
        int collision_count = 0;
    
        const float INTERACTION_RADIUS = FLAMEGPU->environment.getProperty<float>("INTERACTION_RADIUS");
        const float SEPARATION_RADIUS = FLAMEGPU->environment.getProperty<float>("SEPARATION_RADIUS");
        // Iterate location messages, accumulating relevant data and counts.
        for (const auto &message : FLAMEGPU->message_in) {
            // Ignore self messages.
            if (message.getVariable<int>("id") != id) {
                // Get the message location and velocity.
                const float message_x = message.getVariable<float>("x");
                const float message_y = message.getVariable<float>("y");
                const float message_z = message.getVariable<float>("z");
                const float message_fx = message.getVariable<float>("fx");
                const float message_fy = message.getVariable<float>("fy");
                const float message_fz = message.getVariable<float>("fz");
    
                // Check interaction radius
                float separation = vec3Length(agent_x - message_x, agent_y - message_y, agent_z - message_z);
    
                if (separation < (INTERACTION_RADIUS)) {
                    // Update the percieved centre
                    perceived_centre_x += message_x;
                    perceived_centre_y += message_y;
                    perceived_centre_z += message_z;
                    perceived_count++;
    
                    // Update percieved velocity matching
                    global_velocity_x += message_fx;
                    global_velocity_y += message_fy;
                    global_velocity_z += message_fz;
    
                    // Update collision centre
                    if (separation < (SEPARATION_RADIUS)) {  // dependant on model size
                        collision_centre_x += message_x;
                        collision_centre_y += message_y;
                        collision_centre_z += message_z;
                        collision_count += 1;
                    }
                }
            }
        }
    
        // Divide positions/velocities by relevant counts.
        vec3Div(perceived_centre_x, perceived_centre_y, perceived_centre_z, perceived_count);
        vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, perceived_count);
        vec3Div(global_velocity_x, global_velocity_y, global_velocity_z, collision_count);
    
        // Total change in velocity
        float velocity_change_x = 0.f;
        float velocity_change_y = 0.f;
        float velocity_change_z = 0.f;
    
        // Rule 1) Steer towards perceived centre of flock (Cohesion)
        float steer_velocity_x = 0.f;
        float steer_velocity_y = 0.f;
        float steer_velocity_z = 0.f;
        if (perceived_count > 0) {
            const float STEER_SCALE = FLAMEGPU->environment.getProperty<float>("STEER_SCALE");
            steer_velocity_x = (perceived_centre_x - agent_x) * STEER_SCALE;
            steer_velocity_y = (perceived_centre_y - agent_y) * STEER_SCALE;
            steer_velocity_z = (perceived_centre_z - agent_z) * STEER_SCALE;
        }
        velocity_change_x += steer_velocity_x;
        velocity_change_y += steer_velocity_y;
        velocity_change_z += steer_velocity_z;
    
        // Rule 2) Match neighbours speeds (Alignment)
        float match_velocity_x = 0.f;
        float match_velocity_y = 0.f;
        float match_velocity_z = 0.f;
        if (collision_count > 0) {
            const float MATCH_SCALE = FLAMEGPU->environment.getProperty<float>("MATCH_SCALE");
            match_velocity_x = global_velocity_x * MATCH_SCALE;
            match_velocity_y = global_velocity_y * MATCH_SCALE;
            match_velocity_z = global_velocity_z * MATCH_SCALE;
        }
        velocity_change_x += match_velocity_x;
        velocity_change_y += match_velocity_y;
        velocity_change_z += match_velocity_z;
    
        // Rule 3) Avoid close range neighbours (Separation)
        float avoid_velocity_x = 0.0f;
        float avoid_velocity_y = 0.0f;
        float avoid_velocity_z = 0.0f;
        if (collision_count > 0) {
            const float COLLISION_SCALE = FLAMEGPU->environment.getProperty<float>("COLLISION_SCALE");
            avoid_velocity_x = (agent_x - collision_centre_x) * COLLISION_SCALE;
            avoid_velocity_y = (agent_y - collision_centre_y) * COLLISION_SCALE;
            avoid_velocity_z = (agent_z - collision_centre_z) * COLLISION_SCALE;
        }
        velocity_change_x += avoid_velocity_x;
        velocity_change_y += avoid_velocity_y;
        velocity_change_z += avoid_velocity_z;
    
        // Global scale of velocity change
        vec3Mult(velocity_change_x, velocity_change_y, velocity_change_z, FLAMEGPU->environment.getProperty<float>("GLOBAL_SCALE"));
    
        // Update agent velocity
        agent_fx += velocity_change_x;
        agent_fy += velocity_change_y;
        agent_fz += velocity_change_z;
    
        // Bound velocity
        float agent_fscale = vec3Length(agent_fx, agent_fy, agent_fz);
        if (agent_fscale > 1) {
            vec3Div(agent_fx, agent_fy, agent_fz, agent_fscale);
        }
    
        // Apply the velocity
        const float TIME_SCALE = FLAMEGPU->environment.getProperty<float>("TIME_SCALE");
        agent_x += agent_fx * TIME_SCALE;
        agent_y += agent_fy * TIME_SCALE;
        agent_z += agent_fz * TIME_SCALE;
    
        // Bound position
        clampPosition(agent_x, agent_y, agent_z, FLAMEGPU->environment.getProperty<float>("MIN_POSITION"), FLAMEGPU->environment.getProperty<float>("MAX_POSITION"));
    
        // Update global agent memory.
        FLAMEGPU->setVariable<float>("x", agent_x);
        FLAMEGPU->setVariable<float>("y", agent_y);
        FLAMEGPU->setVariable<float>("z", agent_z);
    
        FLAMEGPU->setVariable<float>("fx", agent_fx);
        FLAMEGPU->setVariable<float>("fy", agent_fy);
        FLAMEGPU->setVariable<float>("fz", agent_fz);
    
        return ALIVE;
    }
    )###";

typedef struct Experiment { 

    Experiment(std::string title, 
        unsigned int initialPopSize, unsigned int finalPopSize, unsigned int popSizeIncrement, 
        unsigned int initialNumSpecies, unsigned int finalNumSpecies, unsigned int numSpeciesIncrement,
        unsigned int repetitions,
        double targetPopDensity,
        bool spatial) {
            this->title = title;
            this->initialPopSize = initialPopSize;
            this->finalPopSize = finalPopSize;
            this->popSizeIncrement = popSizeIncrement;
            this->initialNumSpecies = initialNumSpecies;
            this->finalNumSpecies = finalNumSpecies;
            this->numSpeciesIncrement = numSpeciesIncrement;
            this->repetitions = repetitions;
            this->targetPopDensity = targetPopDensity;
            this->spatial = spatial;
        }
    std::string title;
    unsigned int initialPopSize = 100000;
    unsigned int finalPopSize = 100000;
    unsigned int popSizeIncrement = 100000;
    
    unsigned int initialNumSpecies = 1;
    unsigned int finalNumSpecies = 8;
    unsigned int numSpeciesIncrement = 1;

    unsigned int repetitions = 1;
    double targetPopDensity = 1024;
    bool spatial = true;
} Experiment;

int main(int argc, const char ** argv) {

    unsigned int repetitions = 1;
    Experiment smallFixedPop("Small Fixed Pop", 512, 512, 512, 1, 16, 1, repetitions, 1024, true);
    Experiment smallPops("Small Pops", 128, 1024, 128, 1, 8, 1, repetitions, 1024, true);
    Experiment largePops("Large Pops", 1024, 8192, 1024, 1, 8, 1, repetitions, 1024, true);
    Experiment deviceMaxed("Device Maxed", 100000, 100000, 100000, 1, 8, 1, 1, 4096, true);
    //Experiment sweepPopDensity("Sweep Population Density", 4096, 4096, 4096, 1, 8, 1);

    Experiment smallFixedPopBruteForce("Small Fixed Pop Brute Force", 512, 512, 512, 1, 16, 1, repetitions, 1024, false);
    Experiment smallPopsBruteForce("Small Pops Brute Force", 128, 1024, 128, 1, 8, 1, repetitions, 1024, false);
    Experiment largePopsBruteForce("Large Pops Brute Force", 1024, 1024, 1024, 1, 8, 1, 1, 1024, false);

    std::vector<Experiment> experiments = {largePops, largePopsBruteForce};
    
    for (Experiment experiment : experiments) {
        
        std::cout << std::endl << "Starting experiment: " << experiment.title << std::endl;

        unsigned int initialPopSize = experiment.initialPopSize;
        unsigned int finalPopSize = experiment.finalPopSize;
        unsigned int popSizeIncrement = experiment.popSizeIncrement;
        
        unsigned int initialNumSpecies = experiment.initialNumSpecies;
        unsigned int finalNumSpecies = experiment.finalNumSpecies;
        unsigned int numSpeciesIncrement = experiment.numSpeciesIncrement;

        // Target population density per cubic unit
        double targetPopDensity = experiment.targetPopDensity;

        std::vector<double> serialResults;
        std::vector<double> concurrentResults;
        for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
            for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                serialResults.push_back(0.0);
                concurrentResults.push_back(0.0);
            }
        }
        
        for (unsigned int isConcurrent = 0; isConcurrent <= 1; isConcurrent++) {
            for (unsigned int repetition = 0; repetition < experiment.repetitions; repetition++) {
                unsigned int resultsIndex = 0;
                for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
                    for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                        std::cout << "Staring run with popSize: " << popSize << ", species: " << numSpecies << std::endl;
                        ModelDescription model("Boids_Concurrency");

                        /**
                        * GLOBALS
                        */
                        EnvironmentDescription &env = model.Environment();
                        std::vector<unsigned int> populationSizes;
                        for (unsigned int pops = 0; pops < numSpecies; pops++) {
                            populationSizes.push_back(popSize);
                        }
                        {                          
                            // Target pop density
                            double targetVolume = (double)popSize / targetPopDensity;
                            double sideLength = std::cbrt(targetVolume);
                            double halfSideLength = sideLength / 2.0;
                            //std::cout << "Using side length: " << sideLength << std::endl;


                            // Environment Bounds
                            env.newProperty("MIN_POSITION", static_cast<float>(-halfSideLength));
                            env.newProperty("MAX_POSITION", static_cast<float>(halfSideLength));

                            // Initialisation parameter(s)
                            env.newProperty("MAX_INITIAL_SPEED", 1.0f);
                            env.newProperty("MIN_INITIAL_SPEED", 0.01f);

                            // Interaction radius
                            env.newProperty("INTERACTION_RADIUS", 0.1f);
                            env.newProperty("SEPARATION_RADIUS", 0.005f);

                            // Global Scalers
                            env.newProperty("TIME_SCALE", 0.0005f);
                            env.newProperty("GLOBAL_SCALE", 0.15f);

                            // Rule scalers
                            env.newProperty("STEER_SCALE", 0.65f);
                            env.newProperty("COLLISION_SCALE", 0.75f);
                            env.newProperty("MATCH_SCALE", 1.25f);
                        }

                        {   // Location message
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string messageName = "location";
                                messageName += std::to_string(i);
                                if (experiment.spatial) {
                                    MsgSpatial3D::Description &message = model.newMessage<MsgSpatial3D>(messageName);
                                    // Set the range and bounds.
                                    message.setRadius(env.getProperty<float>("INTERACTION_RADIUS"));
                                    message.setMin(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MIN_POSITION"));
                                    message.setMax(env.getProperty<float>("MAX_POSITION"), env.getProperty<float>("MAX_POSITION"), env.getProperty<float>("MAX_POSITION"));
                                    // A message to hold the location of an agent.
                                    message.newVariable<int>("id");
                                    // X Y Z are implicit.
                                    // message.newVariable<float>("x");
                                    // message.newVariable<float>("y");
                                    // message.newVariable<float>("z");
                                    message.newVariable<float>("fx");
                                    message.newVariable<float>("fy");
                                    message.newVariable<float>("fz");
                                } else {
                                    MsgBruteForce::Description &message = model.newMessage<MsgBruteForce>(messageName);
                                    // A message to hold the location of an agent.
                                    message.newVariable<int>("id");
                                    message.newVariable<float>("x");
                                    message.newVariable<float>("y");
                                    message.newVariable<float>("z");
                                    message.newVariable<float>("fx");
                                    message.newVariable<float>("fy");
                                    message.newVariable<float>("fz");
                                }
                                
                            }
                        }
                        for (unsigned int i = 0; i < populationSizes.size(); i++) {
                            {   // Boid agent
                                std::string agentName("Boid" + std::to_string(i));
                                AgentDescription &agent = model.newAgent(agentName);
                                agent.newVariable<int>("id");
                                agent.newVariable<float>("x");
                                agent.newVariable<float>("y");
                                agent.newVariable<float>("z");
                                agent.newVariable<float>("fx");
                                agent.newVariable<float>("fy");
                                agent.newVariable<float>("fz");
                                std::string messageName = "location";
                                messageName += std::to_string(i);
                                std::string outputFuncName = "outputdata";
                                outputFuncName += std::to_string(i);
                                std::string inputFuncName = "inputdata";
                                inputFuncName += std::to_string(i);
                                
                                // Use brute force or spatial?
                                const char* outputFuncSource;
                                const char* inputFuncSource;
                                if (experiment.spatial) {
                                    outputFuncSource = outputdata;
                                    inputFuncSource = inputdata;
                                } else {
                                    outputFuncSource = outputdataBruteForce;
                                    inputFuncSource = inputdataBruteForce;
                                }
                                agent.newRTCFunction(agentName + outputFuncName, outputFuncSource).setMessageOutput(messageName);
                                agent.newRTCFunction(agentName + inputFuncName, inputFuncSource).setMessageInput(messageName);
                            }
                        }

                        /**
                        * Control flow
                        */     
                        {   // Layer #1
                            LayerDescription &layer = model.newLayer();
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                std::string outputFuncName = "outputdata";
                                outputFuncName += std::to_string(i);
                                layer.addAgentFunction(agentName, agentName + outputFuncName);
                            }
                        }
                        {   // Layer #2
                            LayerDescription &layer = model.newLayer();
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                std::string inputFuncName = "inputdata";
                                inputFuncName += std::to_string(i);
                                layer.addAgentFunction(agentName, agentName + inputFuncName);
                            }
                        }


                        /**
                        * Create Model Runner
                        */
                        CUDASimulation cuda_model(model);

                        /**
                        * Create visualisation
                        */
                #ifdef VISUALISATION
                        ModelVis &visualisation = cuda_model.getVisualisation();
                        {
                            float envWidth = env.getProperty<float>("MAX_POSITION") - env.getProperty<float>("MIN_POSITION");
                            const float INIT_CAM = env.getProperty<float>("MAX_POSITION") * 1.25f;
                            visualisation.setInitialCameraLocation(INIT_CAM, INIT_CAM, INIT_CAM);
                            visualisation.setCameraSpeed(0.002f * envWidth);
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                auto &circ_agt = visualisation.addAgent(agentName);
                                // Position vars are named x, y, z; so they are used by default
                                circ_agt.setModel(Stock::Models::ICOSPHERE);
                                circ_agt.setModelScale((i+1)*env.getProperty<float>("SEPARATION_RADIUS"));
                            }
                        }
                        visualisation.activate();
                #endif

                        // Initialisation
                        cuda_model.initialise(argc, argv);

                        // If no xml model file was is provided, generate a population.
                        if (cuda_model.getSimulationConfig().input_file.empty()) {
                            // Set number of steps
                            cuda_model.SimulationConfig().steps = 1000;

                            // Uniformly distribute agents within space, with uniformly distributed initial velocity.
                            std::mt19937 rngEngine(cuda_model.getSimulationConfig().random_seed);
                            std::uniform_real_distribution<float> position_distribution(env.getProperty<float>("MIN_POSITION"), env.getProperty<float>("MAX_POSITION"));
                            std::uniform_real_distribution<float> velocity_distribution(-1, 1);
                            std::uniform_real_distribution<float> velocity_magnitude_distribution(env.getProperty<float>("MIN_INITIAL_SPEED"), env.getProperty<float>("MAX_INITIAL_SPEED"));
                            
                            unsigned int agentCounter = 0;
                            for (unsigned int i = 0; i < populationSizes.size(); i++) {
                                std::string agentName = "Boid";
                                agentName += std::to_string(i);
                                AgentPopulation population(model.Agent(agentName), populationSizes[i]);
                                for (unsigned int j = 0; j < populationSizes[i]; j++) {
                                    AgentInstance instance = population.getNextInstance();
                                    instance.setVariable<int>("id", agentCounter);
                                    agentCounter++;

                                    // Agent position in space
                                    instance.setVariable<float>("x", position_distribution(rngEngine));
                                    instance.setVariable<float>("y", position_distribution(rngEngine));
                                    instance.setVariable<float>("z", position_distribution(rngEngine));

                                    // Generate a random velocity direction
                                    float fx = velocity_distribution(rngEngine);
                                    float fy = velocity_distribution(rngEngine);
                                    float fz = velocity_distribution(rngEngine);
                                    // Generate a random speed between 0 and the maximum initial speed
                                    float fmagnitude = velocity_magnitude_distribution(rngEngine);
                                    // Use the random speed for the velocity.
                                    vec3Normalize(fx, fy, fz);
                                    vec3Mult(fx, fy, fz, fmagnitude);

                                    // Set these for the agent.
                                    instance.setVariable<float>("fx", fx);
                                    instance.setVariable<float>("fy", fy);
                                    instance.setVariable<float>("fz", fz);
                                }
                                cuda_model.setPopulationData(population);
                            }
                        }

                        /**
                        * Execution
                        */
                        cuda_model.CUDAConfig().inLayerConcurrency = isConcurrent;
                        //std::cout << "In layer concurrency set to: " << cuda_model.CUDAConfig().inLayerConcurrency << std::endl;

                        cuda_model.simulate();
                        const auto runTime = cuda_model.getElapsedTimeSimulation();
                        const double averageStepTime = runTime / 1000.0;
                        
                        //std::cout << "Run complete. Average step time: " << averageStepTime << "ms" << std::endl;
                        if (isConcurrent) {
                            concurrentResults[resultsIndex] += averageStepTime;
                        } else {
                            serialResults[resultsIndex] += averageStepTime;
                        }
                        resultsIndex++;

                        /**
                        * Export Pop
                        */
                        // cuda_model.exportData("end.xml");

            #ifdef VISUALISATION
                        visualisation.join();
                        visualisation.close();
            #endif
                    }
                }
            }
        }
        // Compute average results
        for (unsigned int i = 0; i < concurrentResults.size(); i++) {
            concurrentResults[i] /= static_cast<double>(experiment.repetitions);
            serialResults[i] /= static_cast<double>(experiment.repetitions);
        }

        // Output parameters to file
        std::ofstream paramsFile("results/" + experiment.title + "/params.csv");
        if (paramsFile.is_open()) {
            paramsFile << initialPopSize << "," << finalPopSize << "," << popSizeIncrement << std::endl;
            paramsFile << initialNumSpecies << "," << finalNumSpecies << "," << numSpeciesIncrement << std::endl;
        }

        // Output concurrent results to file
        std::ofstream concurrentOutputFile("results/" + experiment.title + "/concurrent.csv");
        if (concurrentOutputFile.is_open()) {
            unsigned int i = 0;
            for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
                for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                    std::string separator = numSpecies == finalNumSpecies ? "\n" : ", ";
                    concurrentOutputFile << concurrentResults[i] << separator;
                    i++;
                }
            }
        }

        // Output serial results to file
        std::ofstream serialOutputFile("results/" + experiment.title + "/serial.csv");
        if (serialOutputFile.is_open()) {
            unsigned int i = 0;
            for (unsigned int popSize = initialPopSize; popSize <= finalPopSize; popSize += popSizeIncrement) {
                for (unsigned int numSpecies = initialNumSpecies; numSpecies <= finalNumSpecies; numSpecies+= numSpeciesIncrement) {
                    std::string separator = numSpecies == finalNumSpecies ? "\n" : ", ";
                    serialOutputFile << serialResults[i] << separator;
                    i++;
                }
            }
        }
    }
    return 0;
}


